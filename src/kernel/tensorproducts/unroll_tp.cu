#include "tensorproducts.hpp"

#include <iostream>
#include <fstream>
#include "hip/hip_runtime.h"
#include "gpu_util.hpp"
#include "jit.hpp"

using namespace std;

UnrollTPImpl::UnrollTPImpl(
    RepTriple &reps,
    std::string jit_kernel,
    KernelLaunchConfig &forward_config_i,
    KernelLaunchConfig &backward_config_i) :
        GenericTensorProductImpl(reps),
        jit(jit_kernel),
        forward_config(forward_config_i),  
        backward_config(backward_config_i) {
    vector<string> kernels = {"loop_unroll_many_to_one", "loop_unroll_backward"};
    jit.compile(kernels, {{}, {}}); 

    if(forward_config.smem > 0) {
        jit.set_max_smem(0, forward_config.smem);
    }

    if(backward_config.smem > 0) {
        jit.set_max_smem(1, backward_config.smem);
    }
}

void UnrollTPImpl::exec_tensor_product(
    uint64_t num_products,
    float* L1_in,
    float* L2_in,
    float* L3_out,
    float* weights) {

    void *args[] = { &num_products, &L1_in, &L2_in, &L3_out }; 
    jit.execute(0, forward_config.num_blocks, forward_config.num_threads, args, forward_config.smem);
}

void UnrollTPImpl::backward(
        size_t num_products,
        float* L1_in, float* L1_grad,
        float* L2_in, float* L2_grad,
        float* weight, float* weight_grad,
        float* L3_grad) {

    void *args[] = { &num_products, &L1_in, &L1_grad, &L2_in, &L2_grad, &weight, &weight_grad, &L3_grad}; 
    jit.execute(1, backward_config.num_blocks, backward_config.num_threads, args, backward_config.smem);
} 

