#include "tensorproducts.hpp"

#include <iostream>
#include <fstream>
#include "hip/hip_runtime.h"
#include "gpu_util.hpp"
#include "jit.hpp"

using namespace std;

JITTPImpl::JITTPImpl(
    std::string jit_kernel,
    KernelLaunchConfig &forward_config_i,
    KernelLaunchConfig &backward_config_i) :
        jit(jit_kernel),
        forward_config(forward_config_i),  
        backward_config(backward_config_i) {
    vector<string> kernels = {"forward", "backward"};
    jit.compile(kernels, {{}, {}}); 

    if(forward_config.smem > 0) {
        jit.set_max_smem(0, forward_config.smem);
    }

    if(backward_config.smem > 0) {
        jit.set_max_smem(1, backward_config.smem);
    }
}

void JITTPImpl::exec_tensor_product(
    uint64_t num_products,
    void* L1_in,
    void* L2_in,
    void* L3_out,
    void* weights) {

    void *args[] = { &num_products, &L1_in, &L2_in, &L3_out, &weights};
    jit.execute(0, forward_config.num_blocks, forward_config.num_threads, args, forward_config.smem);
}

void JITTPImpl::backward(
        size_t num_products,
        void* L1_in, void* L1_grad,
        void* L2_in, void* L2_grad,
        void* weight, void* weight_grad,
        void* L3_grad) {

    void *args[] = { &num_products, &L1_in, &L1_grad, &L2_in, &L2_grad, &weight, &weight_grad, &L3_grad}; 
    jit.execute(1, backward_config.num_blocks, backward_config.num_threads, args, backward_config.smem);
}