#include "tensorproducts.hpp"

#include <iostream>
#include <fstream>
#include "hip/hip_runtime.h"
#include "gpu_util.hpp"
#include "jit.hpp"

using namespace std;

MultTPImpl::MultTPImpl(
    RepTriple &reps,
    std::string jit_kernel,
    KernelLaunchConfig &forward_config_i,
    KernelLaunchConfig &backward_config_i) :
        GenericTensorProductImpl(reps),
        jit(jit_kernel),
        forward_config(forward_config_i),  
        backward_config(backward_config_i) {
    vector<string> kernels = {"forward_kernel", "backward_kernel"};
    jit.compile(kernels, { {}, {} }); 

    if(forward_config.smem > 0) {
        jit.set_max_smem(0, forward_config.smem);
    }

    if(backward_config.smem > 0) {
        jit.set_max_smem(1, backward_config.smem);
    }
}

void MultTPImpl::exec_tensor_product(
    uint64_t num_products,
    float* L1_in,
    float* L2_in,
    float* L3_out,
    float* weights) {

    void *args[] = { &num_products, &L1_in, &L2_in, &L3_out, &weights}; 
    jit.execute(0, forward_config.num_blocks, forward_config.num_threads, args, forward_config.smem);
}

void MultTPImpl::backward(
        size_t num_products,
        float* L1_in, float* L1_grad,
        float* L2_in, float* L2_grad,
        float* weight, float* weight_grad,
        float* L3_grad) {

    void *args[] = { &num_products, &L1_in, &L1_grad, &L2_in, &L2_grad, &weight, &weight_grad, &L3_grad}; 
    jit.execute(1, backward_config.num_blocks, backward_config.num_threads, args, backward_config.smem);
} 

