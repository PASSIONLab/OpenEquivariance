#include "hip/hip_runtime.h"
#include "tensorproducts.hpp"

#include <iostream>
#include "hip/hip_runtime.h"
#include "gpu_util.hpp"

#define THREADS_PER_WARP 32
#define THREAD_BLOCK_SIZE 512

#define A100_SMS 108

using namespace std;

struct Linfo {
    float* ptr;
    uint32_t stride; // Assume here that stride is equal to length of row 
};

template <int MAX_LANE_LENGTH, int REDUCTION_DEPTH>
__global__ void shuffle_tp_kernel(
    size_t num_products,
    Linfo L1,
    Linfo L2,
    Linfo L3,
    float* warp_values_ptr,
    int* l1_indices_ptr,
    int* l2_indices_ptr,
    int* red_lanes_ptr) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int warp_id = idx / THREADS_PER_WARP;
    int lane_id = idx % THREADS_PER_WARP;

    float values[MAX_LANE_LENGTH];
    int l1_indices[MAX_LANE_LENGTH];
    int l2_indices[MAX_LANE_LENGTH];
    int red_lanes[REDUCTION_DEPTH];

    // Load values into registers from global memory
    for(int i = 0; i < MAX_LANE_LENGTH; i++) {
        values[i] = warp_values_ptr[i * THREADS_PER_WARP + lane_id];
        l1_indices[i] = l1_indices_ptr[i * THREADS_PER_WARP + lane_id];
        l2_indices[i] = l2_indices_ptr[i * THREADS_PER_WARP + lane_id];
    }

    for(int i = 0; i < REDUCTION_DEPTH; i++) {
        red_lanes[i] = red_lanes_ptr[i * THREADS_PER_WARP + lane_id];
    }

    size_t warps_launched = blockDim.x * gridDim.x / 16;
    size_t nnz_per_warp = (num_products + warps_launched - 1) / warps_launched;

    size_t start = warp_id * nnz_per_warp;
    size_t end = min(start + nnz_per_warp, num_products);

    for(size_t i = start; i < end; i++) {
        float l1_vec = 0.0;
        float l2_vec = 0.0;
        float l3_vec = 0.0;

        // Step 1: Load vectors into warp lanes 
        if(lane_id < L1.stride) {
            float* l1_start = L1.ptr + i * L1.stride;
            l1_vec = l1_start[lane_id];
        }
        if(lane_id < L2.stride) {
            float* l2_start = L2.ptr + i * L2.stride;
            l2_vec = l2_start[lane_id];
        }

        // Step 2: Shuffle and multiply
        #pragma unroll
        for(int j = 0; j < MAX_LANE_LENGTH; j++) {
            float l1_val = __shfl_sync(0xFFFFFFFF, l1_vec, l1_indices[j]);
            float l2_val = __shfl_sync(0xFFFFFFFF, l2_vec, l2_indices[j]);
            l3_vec += l1_val * l2_val * values[j]; // TODO: Can have multiple accumulators 
        }

        // Step 3: Reduce if necessary
        #pragma unroll
        for(int j = 0; j < REDUCTION_DEPTH; j++) {
            float bcast_value = lane_id == 0 ? 0.0 : l3_vec;
            l3_vec += __shfl_sync(0xFFFFFFFF, bcast_value, red_lanes[j]); 
        }

        // Step 4: Store back 
        if(lane_id < L3.stride) {
            float* l3_start = L3.ptr + i * L3.stride;
            l3_start[lane_id] = l3_vec;
        }
    }
}

void ShuffleTensorProductImpl::exec_tensor_product(
    uint64_t num_products,
    float* L1_in,
    float* L2_in,
    float* L3_out) {
        
    // Not really necessary
    gpuErrchk( hipMemset(L3_out, 0.0, L3.get_rep_length() * num_products * sizeof(float)) )

    bool executed_kernel = false;

    if(this->max_lane_length == 4 && this->reduction_depth == 2) {
        executed_kernel = true;
        shuffle_tp_kernel<4, 3>
            <<<A100_SMS, THREAD_BLOCK_SIZE>>>(
                num_products,
                {L1_in, static_cast<uint32_t>(L1.get_rep_length())},
                {L2_in, static_cast<uint32_t>(L2.get_rep_length())},
                {L3_out, static_cast<uint32_t>(L3.get_rep_length())},
                warp_values.ptr,
                l1_indices.ptr,
                l2_indices.ptr,
                red_lanes.ptr
            ); 
    }

    hipDeviceSynchronize();
    gpuErrchk( hipGetLastError() );

    if(!executed_kernel) {
        throw std::runtime_error("Unsupported lane length and reduction depth: " + std::to_string(this->max_lane_length) + ", " + std::to_string(this->reduction_depth));
    }
}
