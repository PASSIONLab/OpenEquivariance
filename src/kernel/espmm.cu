#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "espmm.hpp"
#include <cassert>

#DEFINE THREADS_PER_WARP 32

using namespace std;

/*
* This is a naive version of the code that uses atomics
* to perform the accumulation. Proof of concept to test
* the shuffle-add engine. 
*/

__global__ void espmm_v1(
    ESPMM_Context ctx,
    uint64_t edge_count,
    uint64_t* rows,
    uint64_t* cols,
    float* X_in,
    float* edge_features,
    float* X_out
    ) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int global_warp_idx  = idx / THREADS_PER_WARP;
    int lane_id = idx % THREADS_PER_WARP;

    if(global_warp_idx < edge_count) {
        uint64_t row = rows[global_warp_idx];
        uint64_t col = cols[global_warp_idx];

        float X_in_val = X_in[col * ctx.X_in_row_len + lane_id]; 
        X_out[row * ctx.X_out_row_len + lane_id] += X_in_val; 



    }

}


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void check_cuda_device() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    if(nDevices == 0) {
        cout << "Error, no CUDA-capable device detected!" << endl;
        exit(1);
    }
}

void equivariant_spmm_cpu(
        ESPMM_Context &context,
        uint64_t edge_count,
        uint64_t* rows,
        uint64_t* cols,
        float* X_in,
        float* X_out,
        float* edge_features) {

    check_cuda_device();

    uint64_t *d_rows, *d_cols;
    double *d_X_in, *d_edge_features, *d_X_out;

    gpuErrchk( hipMalloc((void**)&d_rows, edge_count * sizeof(uint64_t)))
    gpuErrchk( hipMalloc((void**)&d_cols, edge_count * sizeof(uint64_t)))
    gpuErrchk( hipMalloc((void**)&d_X_in, context.node_count * context.X_in_row_len * sizeof(float)))
    gpuErrchk( hipMalloc((void**)&d_edge_features, edge_count * context.edge_rowlen * sizeof(float)))
    gpuErrchk( hipMalloc((void**)&d_X_out, context.node_count * context.X_out_row_len * sizeof(float))) 

    gpuErrchk( hipMemcpy(d_rows, rows, edge_count * sizeof(uint64_t), hipMemcpyHostToDevice));
    gpuErrchk( hipMemcpy(d_cols, cols, edge_count * sizeof(uint64_t), hipMemcpyHostToDevice));
    gpuErrchk( hipMemcpy(d_X_in, X_in, context.node_count * context.X_in_row_len * sizeof(float), hipMemcpyHostToDevice))
    gpuErrchk( hipMemcpy(d_edge_features, edge_features, edge_count * context.edge_rowlen * sizeof(float), hipMemcpyHostToDevice))
    gpuErrchk( hipMemset(d_X_out, 0, context.X_out_row_len * context.node_count * sizeof(float)))

    cout << "Computation goes here!" << endl;

    hipMemcpy(X_out, d_X_out, context.node_count * context.X_out_row_len * sizeof(float), hipMemcpyDeviceToHost);

    gpuErrchk( hipFree(d_rows))
    gpuErrchk( hipFree(d_cols))
    gpuErrchk( hipFree(d_X_in))
    gpuErrchk( hipFree(d_edge_features))
    gpuErrchk( hipFree(d_X_out))
}
