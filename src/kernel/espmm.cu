#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "espmm.hpp"
#include <cassert>

using namespace std;

__global__ void espmm(
    uint64_t edge_count,
    uint64_t* rows,
    uint64_t* cols,
    float* X_in,
    float* X_out) {

}


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

uint64_t feature_length(uint64_t L) {
    return 2 * L + 1;
}

void check_cuda_device() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    if(nDevices == 0) {
        cout << "Error, no CUDA-capable device detected!" << endl;
        exit(1);
    }
}

void equivariant_spmm_cpu(
        ESPMM_Context &context,
        uint64_t edge_count,
        uint64_t* rows,
        uint64_t* cols,
        float* X_in,
        float* X_out,
        float* edge_features) {

    check_cuda_device();

    uint64_t *d_rows, *d_cols;
    double *d_X_in, *d_edge_features, *d_X_out;

    gpuErrchk( hipMalloc((void**)&d_rows, edge_count * sizeof(uint64_t)))
    gpuErrchk( hipMalloc((void**)&d_cols, edge_count * sizeof(uint64_t)))
    gpuErrchk( hipMalloc((void**)&d_X_in, context.node_count * feature_length(context.L1) * sizeof(float)))
    gpuErrchk( hipMalloc((void**)&d_edge_features, edge_count * feature_length(context.L2) * sizeof(float)))
    gpuErrchk( hipMalloc((void**)&d_X_out, context.node_count * feature_length(context.L3) * sizeof(float))) 

    gpuErrchk( hipMemcpy(d_rows, rows, edge_count * sizeof(uint64_t), hipMemcpyHostToDevice));
    gpuErrchk( hipMemcpy(d_cols, cols, edge_count * sizeof(uint64_t), hipMemcpyHostToDevice));
    gpuErrchk( hipMemcpy(d_X_in, X_in, context.node_count * feature_length(context.L1) * sizeof(float), hipMemcpyHostToDevice))
    gpuErrchk( hipMemcpy(d_edge_features, edge_features, edge_count * feature_length(context.L2) * sizeof(float), hipMemcpyHostToDevice))

    cout << "Computation goes here!" << endl;

    hipMemcpy(X_out, d_X_out, context.node_count * feature_length(context.L3) * sizeof(float), hipMemcpyDeviceToHost);

    gpuErrchk( hipFree(d_rows))
    gpuErrchk( hipFree(d_cols))
    gpuErrchk( hipFree(d_X_in))
    gpuErrchk( hipFree(d_edge_features))
    gpuErrchk( hipFree(d_X_out))
}
