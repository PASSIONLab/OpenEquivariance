#include <iostream>
#include <hip/hip_runtime.h>
#include "espmm.hpp"
#include <cassert>

using namespace std;

// This function accepts CPU pointers and copies
// its data to the GPU. 
/*void equivariant_spmm_cpu(
uint64_t node_count,
uint64_t edge_count,
uint64_t L1, 
uint64_t L2,
uint64_t L3,
uint64_t* row_ptr,
uint64_t* cols,
double* X_in,
double* X_out,
double* edge_features) {

}*/

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


uint64_t feature_length(uint64_t L) {
    return 2 * L + 1;
}

void check_cuda_device() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    if(nDevices == 0) {
        cout << "Error, no CUDA-capable device detected!" << endl;
        exit(1);
    }
}

void equivariant_spmm_cpu(
        ESPMM_Context &context,
        uint64_t edge_count,
        uint64_t* rows,
        uint64_t* cols,
        double* X_in,
        double* X_out,
        double* edge_features) {

    check_cuda_device();

    uint64_t *d_rows, *d_cols;
    double *d_X_in, *d_edge_features, *d_X_out;

    gpuErrchk( hipMalloc((void**)&d_rows, edge_count * sizeof(uint64_t)))
    gpuErrchk( hipMalloc((void**)&d_cols, edge_count * sizeof(uint64_t)))
    gpuErrchk( hipMalloc((void**)&d_X_in, context.node_count * feature_length(context.L1) * sizeof(double)))
    gpuErrchk( hipMalloc((void**)&d_edge_features, edge_count * feature_length(context.L2) * sizeof(double)))
    gpuErrchk( hipMalloc((void**)&d_X_out, context.node_count * feature_length(context.L3) * sizeof(double))) 

    gpuErrchk( hipMemcpy(d_rows, rows, edge_count * sizeof(uint64_t), hipMemcpyHostToDevice));
    gpuErrchk( hipMemcpy(d_cols, cols, edge_count * sizeof(uint64_t), hipMemcpyHostToDevice));
    gpuErrchk( hipMemcpy(d_X_in, X_in, context.node_count * feature_length(context.L1) * sizeof(double), hipMemcpyHostToDevice))
    gpuErrchk( hipMemcpy(d_edge_features, edge_features, edge_count * feature_length(context.L2) * sizeof(double), hipMemcpyHostToDevice))

    cout << "Computation goes here!" << endl;

    hipMemcpy(X_out, d_X_out, context.node_count * feature_length(context.L3) * sizeof(double), hipMemcpyDeviceToHost);

    gpuErrchk( hipFree(d_rows))
    gpuErrchk( hipFree(d_cols))
    gpuErrchk( hipFree(d_X_in))
    gpuErrchk( hipFree(d_edge_features))
    gpuErrchk( hipFree(d_X_out))
}
