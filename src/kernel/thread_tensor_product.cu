#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include "espmm.hpp"
#include "gpu_util.hpp"

#define THREADS_PER_WARP 32
#define THREAD_BLOCK_SIZE 1024

using namespace std;

__global__ void thread_tp_kernel(
        size_t num_products,
        float* L1_in,
        size_t L1_stride,
        float* L2_in,
        size_t L2_stride,
        float* L3_out,
        size_t L3_stride,

        size_t nnz,
        uint8_t* coord1, 
        uint8_t* coord2, 
        uint8_t* coord3,
        float* values) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < num_products) {
        float* L1_vec = L1_in + (idx * L1_stride);
        float* L2_vec = L2_in + (idx * L2_stride);
        float* L3_vec = L3_out + (idx * L3_stride);

        for(int i = 0; i < nnz; i++) {
            L3_vec[coord3[i]] += L1_vec[coord1[i]] * L2_vec[coord2[i]] * values[i];
        }
    }
}

void ThreadTensorProductImpl::exec_tensor_product(
        uint64_t num_products,
        float* L1_in,
        float* L2_in,
        float* L3_out) {

    size_t L1_stride = get_row_length(1);
    size_t L2_stride = get_row_length(2);
    size_t L3_stride = get_row_length(3);

    gpuErrchk( hipMemset(L3_out, 0.0, L3_stride * num_products * sizeof(float)) ) 
    size_t nnz = values.size;

    thread_tp_kernel<<<round_up(num_products, THREAD_BLOCK_SIZE) / THREAD_BLOCK_SIZE, THREAD_BLOCK_SIZE>>>(
            num_products, 
            L1_in,
            L1_stride,
            L2_in,
            L2_stride,
            L3_out,
            L3_stride,

            nnz,
            coord1.ptr,
            coord2.ptr,
            coord3.ptr,
            values.ptr); 

    gpuErrchk( hipGetLastError() );
}
