#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__ void add(int *a, int *b, int *c) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    c[idx] = a[idx] + b[idx];
}

int main() {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int n = 10;

    // Allocate host memory
    a = (int*)malloc(n * sizeof(int));
    b = (int*)malloc(n * sizeof(int));
    c = (int*)malloc(n * sizeof(int));

    // Initialize host memory
    for (int i = 0; i < n; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Allocate device memory
    hipMalloc((void**)&d_a, n * sizeof(int));
    hipMalloc((void**)&d_b, n * sizeof(int));
    hipMalloc((void**)&d_c, n * sizeof(int));

    // Copy host memory to device
    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    int blockSize = 4;
    int numBlocks = (n + blockSize - 1) / blockSize;
    add<<<numBlocks, blockSize>>>(d_a, d_b, d_c);

    // Copy device memory to host
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

    // Print result
    for (int i = 0; i < n; i++) {
        std::cout << "c[" << i << "] = " << c[i] << std::endl;
    }

    // Free memory
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}